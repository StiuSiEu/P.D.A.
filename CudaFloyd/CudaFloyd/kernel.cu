#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define N 5

__global__ void RoyFloyd(int a[N][N], int k)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	if (a[i][j] > a[i][k] + a[k][j])
		a[i][j] = a[i][k] + a[k][j];
}


int main()
{
	int h_a[N][N] = {
		{ 0,   2, 99, 10 ,99 },
		{ 2,   0,  3, 99 ,99 },
		{ 99,   3,  0,  1 , 8 },
		{ 10,  99,  1,  0 ,99 },
		{ 99,  99,  8, 99 , 0 }
	};

	int h_c = 0;
	size_t size = N * N * sizeof(int);
	int *d_a;
	//int *d_c;
	hipMalloc(&d_a, size);
//	hipMalloc(&d_c, sizeof(int));

	int numBlocks = 1;
	dim3 threadsPerBlock(N, N);

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);


	for (h_c = 0; h_c < N; h_c++) {
		//hipMemcpy(d_c, &h_c, sizeof(int), hipMemcpyHostToDevice);
		RoyFloyd << <1, threadsPerBlock >> > (d_a, h_c);
	}

	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);

	for (int i = 0;i <= N;i++)
	{
		for (int j = 0;j <= N;j++)
		{
			printf("%d ", h_a[i][j]);
		}
		printf("\n");
	}
	hipFree(d_a);
//	hipFree(d_c);
	return 0;
}
