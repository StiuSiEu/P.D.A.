#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__device__ int get_index_to_check(int thread, int num_threads, int set_size, int offset) {

	// Integer division trick to round up
	return (((set_size + num_threads) / num_threads) * thread) + offset;
}

// Device code
__global__ void p_ary_search(int search, int array_length, int *arr, int *ret_val)
{
	const int num_threads = blockDim.x * gridDim.x;
	const int thread = blockIdx.x * blockDim.x + threadIdx.x;
	int set_size = array_length;
	
	ret_val[0] = -1; 
	ret_val[1] = 0; 

	while (set_size != 0)
	{
		int offset = ret_val[1];

		__syncthreads();

		// Get the next index to check
		int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);

		// If the index is outside the bounds of the array do not check it
		if (index_to_check < array_length)
		{
			// If the next index is outside the bounds of the array, then set it to maximum array size
			int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);
				if (next_index_to_check >= array_length)
				{
					next_index_to_check = array_length - 1;
				}

				// If we're at the mid section of the array reset the offset to this index
				if (search > arr[index_to_check] && (search < arr[next_index_to_check]))
				{
					ret_val[1] = index_to_check;
				}
				else if (search == arr[index_to_check])
				{
					// Set the return var if find it
					ret_val[0] = index_to_check;
				}
		}

		// Since this is a paralel array search divide by our total threads to get the next set size
		set_size = set_size / num_threads;

		// Sync up so no threads jump ahead
		__syncthreads();
	}
}

// Host code
int main()
{
	int N = 23;
	size_t size = N * sizeof(int);
	
	int search=5;
	
	//int h_result;
	int *h_result = (int*)malloc(sizeof(int) * 2);
	int *d_result;
	hipMalloc((void**)&d_result, sizeof(int) * 2);

	hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

	// Allocate input vector h_A in host memory
	int* h_A = (int*)malloc(size);
	// Initialize input vectors
	for (int i = 0;i < N;i++)
	{
		h_A[i] = i;
	}
	// Allocate vector in device memory
	int* d_A;
	hipMalloc(&d_A, size);
	
	// Copy vectors from host memory to device memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);


	// Invoke kernel
	p_ary_search << <16, 64 >> >(search, N, d_A, d_result);
	

	// Copy result from device memory to host memory h_C contains the result in host memory
	hipMemcpy(h_result, d_result, 2 * sizeof(int), hipMemcpyDeviceToHost);
	
	
	printf("the result is :%d", h_result[0]);

	// Free device memory
	/*hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	*/
	scanf("%d", N);
}

